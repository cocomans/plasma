#include "hip/hip_runtime.h"
#include "ParticleListGPU.cuh"
#include "ChargeTally.h"
#include "StressTally.h"
#include "CurrentTallyCPU.h"
#include "CurrentTallyGPU.cuh"
#include "HOMoments.h"
#include "PlasmaData.h"
#include "ProblemInitializer.h"
#include "FieldDataGPU.cuh"
#include "math.h"
#include "omp.h"
#include "Util/GPU_Utils.h"
#include "ClusterInfo.cuh"
#include "PExitCheck.h"
#include "ParticleObjNT.h"



__constant__ PlasmaData pdata_c;
__constant__ HOMoments moments_c;

//FieldDataGPU* fields_gpu;
//__constant__ ParticleListGPU* particlesGPU_c;

const PlasmaData* pdata_c_address = &pdata_c;
const HOMoments* moments_c_address = &moments_c;
//const FieldDataGPU* fields_c_address = &fields_c;
//const ParticleListGPU* particlesGPU_c_address = &particlesGPU_c;

__host__ __device__ __attribute__((noinline))
ParticleListGPU::ParticleListGPU()
{
	device_type = 1;
}

__host__ __device__ __attribute__((noinline))
ParticleListGPU::~ParticleListGPU()
{
	/*
	// Free realkind arrays
	for(int i=0;i<ParticleList_nfloats;i++)
	{
		free(*get_float(i));
	}

	// Allocate int arrays
	for(int i=0;i<ParticleList_nints;i++)
	{
		free(*get_int(i));
	}

	// allocate short ints for cluster id's
	free(cluster_id);
	*/
}

void ParticleListGPU::copy_from(const ParticleList* list_in)
{
	ispecies = list_in -> ispecies;

	enum hipMemcpyKind kind;

	if(list_in->device_type != device_type)
	{
		if(list_in->device_type != 1)
			kind = hipMemcpyHostToDevice;
		else
			kind = hipMemcpyDeviceToHost;
	}
	else
	{
		if(device_type == 1)
			kind = hipMemcpyDeviceToDevice;
		else
			kind = hipMemcpyHostToHost;
	}
	// Free realkind arrays
	for(int i=0;i<ParticleList_nfloats;i++)
	{
		CUDA_SAFE_CALL(hipMemcpyAsync(*get_float(i),*(list_in->get_float(i)),nptcls*sizeof(realkind),kind));
	}

	// Allocate int arrays
	for(int i=0;i<ParticleList_nints;i++)
	{
		CUDA_SAFE_CALL(hipMemcpyAsync(*get_int(i),*(list_in->get_int(i)),nptcls*sizeof(int),kind));
	}

	// allocate short ints for cluster id's
	CUDA_SAFE_CALL(hipMemcpyAsync(cluster_id,(list_in->cluster_id),nptcls*sizeof(int),kind));
	CUDA_SAFE_CALL(hipMemcpyAsync(num_subcycles,(list_in->num_subcycles),nptcls*sizeof(int),kind));
//	CUDA_SAFE_CALL(hipMemcpyAsync(particleIDs,(list_in->particleIDs),nptcls*sizeof(int),kind));
//	CUDA_SAFE_CALL(hipMemcpyAsync(particleIDs_original,(list_in->particleIDs_original),nptcls*sizeof(int),kind));

	CUDA_SAFE_CALL(hipDeviceSynchronize());

}


void ParticleListGPU::allocate(PlasmaData* pdata,int nptcls_in)
{
	device_type = 1;
	// Allocate memory for particles
	nptcls_allocated = nptcls_in;

	nptcls = nptcls_in;

	//plot = gnuplot_init();

	// Allocate realkind arrays
	for(int i=0;i<ParticleList_nfloats;i++)
	{
		CUDA_SAFE_CALL(hipMalloc((void**)get_float(i),nptcls_allocated*sizeof(realkind)));
	}

	// Allocate int arrays
	for(int i=0;i<ParticleList_nints;i++)
	{
		CUDA_SAFE_CALL(hipMalloc((void**)get_int(i),nptcls_allocated*sizeof(int)));
	}

	// allocate short ints for cluster id's
	CUDA_SAFE_CALL(hipMalloc((void**)&cluster_id,nptcls_allocated*sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&num_subcycles,nptcls_allocated*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&num_piccard,nptcls_allocated*sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc((void**)&num_piccard2,nptcls_allocated*sizeof(double)));

	CUDA_SAFE_CALL(hipMalloc((void**)&nsubcycles_current,nptcls_allocated*sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&particleIDs,nptcls_allocated*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&particleIDs_original,nptcls_allocated*sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&buffer64,nptcls_allocated*sizeof(realkind)));
	CUDA_SAFE_CALL(hipMalloc((void**)&buffer32,nptcls_allocated*sizeof(int)));

	CUDA_SAFE_CALL(hipMemset(num_piccard,0,nptcls*sizeof(double)));
	CUDA_SAFE_CALL(hipMemset(num_piccard2,0,nptcls*sizeof(double)));

	gridsize = 6*16;
	blocksize = 256;

	gridsize = min(gridsize,(nptcls+blocksize-1)/blocksize);

	CUDA_SAFE_CALL(hipMalloc((void**)&nsubcycles_thread,gridsize*blocksize*sizeof(int)));

	int nclusters_x = (pdata->nx+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
	int nclusters_y = (pdata->ny+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
	int nclusters_z = (pdata->nz+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;

	nclusters = nclusters_x*nclusters_y*nclusters_z;
	printf("nclusters = %i\n",nclusters);

	CUDA_SAFE_CALL(hipMalloc((void**)&clusters,nclusters*sizeof(ClusterInfo)));



	CUDA_SAFE_CALL(hipGetSymbolAddress((void**)&pdata_d,pdata_c));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pdata_c),pdata,sizeof(PlasmaData)));

//	CUDA_SAFE_CALL(hipGetSymbolAddress((void**)&this_cp,particlesGPU_c));
	CUDA_SAFE_CALL(hipGetSymbolAddress((void**)&moments_d_cp,moments_c));
//	CUDA_SAFE_CALL(hipGetSymbolAddress((void**)&fields_d_cp,fields_c));

	CUDA_SAFE_CALL(hipMalloc((void**)&this_cp,sizeof(ParticleListGPU)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fields_d_cp,sizeof(FieldDataGPU)));

	moments_d = new HOMoments(pdata,1);
	fields_d = new FieldDataGPU();
	fields_d -> allocate(pdata);

}

void ParticleListGPU::init(ProblemInitializer* initializer, HOMoments* moments)
{

	CurrentTallyCPU currents(&moments->get_val(0,0,0,ispecies,HOMoments_currentx),
						  &moments->get_val(0,0,0,ispecies,HOMoments_currenty),
						  &moments->get_val(0,0,0,ispecies,HOMoments_currentz),
						  make_int3(moments->pdata->nx,moments->pdata->ny,moments->pdata->nz),
						  moments->pdata->dxdi,moments->pdata->dydi,moments->pdata->dzdi,
						  moments->pdata->ndimensions);

	ChargeTally charge(&moments->get_val(0,0,0,ispecies,HOMoments_charge),
						  make_int3(moments->pdata->nx,moments->pdata->ny,moments->pdata->nz),
						  moments->pdata->dxdi,moments->pdata->dydi,moments->pdata->dzdi,
						  moments->pdata->ndimensions);

	StressTally stress(&moments->get_val(0,0,0,ispecies,HOMoments_S2xx),
			&moments->get_val(0,0,0,ispecies,HOMoments_S2xy),
			&moments->get_val(0,0,0,ispecies,HOMoments_S2xz),
			&moments->get_val(0,0,0,ispecies,HOMoments_S2yy),
			&moments->get_val(0,0,0,ispecies,HOMoments_S2yz),
			&moments->get_val(0,0,0,ispecies,HOMoments_S2zz),
						  moments->pdata->nx,moments->pdata->ny,moments->pdata->nz,
						  moments->pdata->ndimensions,moments->pdata->nVelocity);


	moments -> set_vals(0);


	// do this on the CPU

	// allocate temporary space on the cpu
	realkind* pxt = (realkind*)malloc(nptcls*sizeof(realkind));
	realkind* pyt = (realkind*)malloc(nptcls*sizeof(realkind));
	realkind* pzt = (realkind*)malloc(nptcls*sizeof(realkind));
	realkind* vxt = (realkind*)malloc(nptcls*sizeof(realkind));
	realkind* vyt = (realkind*)malloc(nptcls*sizeof(realkind));
	realkind* vzt = (realkind*)malloc(nptcls*sizeof(realkind));

	int* ixt = (int*)malloc(nptcls*sizeof(int));
	int* iyt = (int*)malloc(nptcls*sizeof(int));
	int* izt = (int*)malloc(nptcls*sizeof(int));

	printf("Populating Particle data\n");


	for(int i=0;i<nptcls;i++)
	{
		realkind pxtt,pytt,pztt,vxtt,vytt,vztt;
		int ixtt,iytt,iztt;


		initializer->init_particle(pxtt,pytt,pztt,ixtt,iytt,iztt,vxtt,vytt,vztt,ispecies,i);

		// Set Position Values, ifloat = 0-2
		pxt[i] = pxtt;
		pyt[i] = pytt;
		pzt[i] = pztt;

		// Set Position Index Values, iint = 0-2
		ixt[i] = ixtt;
		iyt[i] = iytt;
		izt[i] = 0;

		// Set Velocity Values, ifloat = 3-5
		vxt[i] = vxtt;
		vyt[i] = vytt;
		vzt[i] = vztt;




	}

	for(int i=0;i<nptcls;i++)
	{
	currents.tally(pxt[i],pyt[i],pzt[i],vxt[i],vyt[i],vzt[i],ixt[i],iyt[i],izt[i],1.0);

	charge.tally(pxt[i],pyt[i],pzt[i],
			ixt[i],iyt[i],izt[i],
			1.0);

	stress.tally(pxt[i],pyt[i],pzt[i],
			vxt[i],vyt[i],vzt[i],
			ixt[i],iyt[i],izt[i],
			1.0);
	}


	printf("Copying particle data to the device\n");

	CUDA_SAFE_CALL(hipMemset(dt_finished,0,nptcls*sizeof(realkind)));



	CUDA_SAFE_CALL(hipMemcpyAsync(px,pxt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(py,pyt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(pz,pzt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMemcpyAsync(vx,vxt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(vy,vyt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(vz,vzt,nptcls*sizeof(realkind),hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMemcpyAsync(ix,ixt,nptcls*sizeof(int),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(iy,iyt,nptcls*sizeof(int),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(iz,izt,nptcls*sizeof(int),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(num_subcycles,0,nptcls*sizeof(int)));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	int blockSize = 512;
	int gridSize = 96;

	printf("Writing particle cluster IDs\n");
	CUDA_SAFE_KERNEL((write_cluster_ids<2,3><<<blockSize,gridSize>>>(
			pdata_d,*this,ifinished,nptcls)));


	//check_int_vals(cluster_id,nptcls);

	printf("Calling Thrust sort\n");
	SortByKey(particleIDs,cluster_id,nptcls);

	//check_int_vals(cluster_id,nptcls);
	ReorderData(particleIDs,nptcls);

	//check_int_vals(cluster_id,nptcls);
	free(pxt);
	free(pyt);
	free(pzt);
	free(vxt);
	free(vyt);
	free(vzt);
	free(ixt);
	free(iyt);
	free(izt);
}

realkind ParticleListGPU::evaluate_energy(PlasmaData* pdata)
{
	double etotal = 0.0;
	for(int i=0;i<nptcls;i++)
	{
//		etotal += get_fvalue(i,3)* get_fvalue(i,3);
//		etotal += get_fvalue(i,4)* get_fvalue(i,4);
//		etotal += get_fvalue(i,5)* get_fvalue(i,5);
	}

	etotal = etotal * pdata->mspecies[ispecies] * 0.5/((double)pdata->nptcls_total);

	return etotal;
}

double4 ParticleListGPU::subcycle_stats(PlasmaData* pdata)
{
	int* num_subcycles_temp = (int*)malloc(nptcls*sizeof(int));

	CUDA_SAFE_CALL(hipMemcpy(num_subcycles_temp,num_subcycles,nptcls*sizeof(int),
			hipMemcpyDeviceToHost));

	double scale = pdata->npiccard_outer;
	double mean = 0;
	double mean2 = 0;
	double mins = num_subcycles_temp[0]/scale;
	double maxs = num_subcycles_temp[0]/scale;
	int imax = 0;
	int imin = 0;
	for(int i=0;i<nptcls;i++)
	{
		if(mins > num_subcycles_temp[i]/scale)
		{
			mins = num_subcycles_temp[i]/scale;
			imin = i;
		}
		if(maxs < num_subcycles_temp[i]/scale)
		{
			maxs = num_subcycles_temp[i]/scale;
			imax = i;
		}

		mean += num_subcycles_temp[i]/((double)nptcls*scale);
		mean2 += num_subcycles_temp[i]*num_subcycles_temp[i]/((double)nptcls*scale*scale);
	}

	double std_diff;

	std_diff = sqrt(fabs(mean*mean - mean2));
	printf("Particle Subcycle Stats:\n");
	printf("Avg Subcycles: %f +/- %f\n",mean,std_diff);
	printf("Min / Max: %f[%i] / %f[%i]\n",mins,imin,maxs,imax);

	free(num_subcycles_temp);

	return make_double4(mean,std_diff,mins,maxs);
}

double4 ParticleListGPU::piccard_stats(PlasmaData* pdata)
{
	int* num_subcycles_temp = (int*)malloc(nptcls*sizeof(int));
	double* num_piccard_temp = (double*)malloc(nptcls*sizeof(double));
	double* num_piccard2_temp = (double*)malloc(nptcls*sizeof(double));

	CUDA_SAFE_CALL(hipMemcpy(num_subcycles_temp,num_subcycles,nptcls*sizeof(int),
			hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipMemcpy(num_piccard_temp,num_piccard,nptcls*sizeof(double),
			hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipMemcpy(num_piccard2_temp,num_piccard2,nptcls*sizeof(double),
			hipMemcpyDeviceToHost));

	double scale = pdata->npiccard_outer;
	double mean = 0;
	double mean2 = 0;
	double mins = num_piccard_temp[0]/num_subcycles_temp[0];
	double maxs = num_piccard_temp[0]/num_subcycles_temp[0];
	int imax = 0;
	int imin = 0;
	for(int i=0;i<nptcls;i++)
	{
		if(mins > num_piccard_temp[i]/num_subcycles_temp[i])
		{
			mins = num_piccard_temp[i]/num_subcycles_temp[i];
			imin = i;
		}
		if(maxs < num_piccard_temp[i]/num_subcycles_temp[i])
		{
			maxs = num_piccard_temp[i]/num_subcycles_temp[i];
			imax = i;
		}

		mean += num_piccard_temp[i]/((double)nptcls*num_subcycles_temp[i]);
		mean2 += num_piccard_temp[i]*num_piccard_temp[i]/((double)nptcls*num_subcycles_temp[i]*num_subcycles_temp[i]);
	}

	double std_diff;

	std_diff = sqrt(fabs(mean*mean - mean2));
	printf("Particle Piccard Stats(GPU):\n");
	printf("Avg Piccard: %f +/- %f\n",mean,std_diff);
	printf("Min / Max: %f[%i] / %f[%i]\n",mins,imin,maxs,imax);

	free(num_subcycles_temp);
	free(num_piccard_temp);
	free(num_piccard2_temp);

	return make_double4(mean,std_diff,mins,maxs);
}



void ParticleListGPU::plot_particles(PlasmaData* pdata)
{
	float* x_vals = (float*)malloc(nptcls*sizeof(float));
	float* y_vals = (float*)malloc(nptcls*sizeof(float));

	realkind* px_t = (realkind*)malloc(nptcls*sizeof(realkind));
	int* ix_t = (int*)malloc(nptcls*sizeof(int));
	realkind* vx_t = (realkind*)malloc(nptcls*sizeof(realkind));

	CUDA_SAFE_CALL(hipMemcpy(px_t,px,nptcls*sizeof(realkind),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(ix_t,ix,nptcls*sizeof(int),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(vx_t,vx,nptcls*sizeof(realkind),hipMemcpyDeviceToHost));

	for(int i=0;i<nptcls;i++)
	{
		x_vals[i] = (px_t[i]+ix_t[i])*pdata->dxdi + pdata->xmin;
		y_vals[i] = vx_t[i];

//		printf("particle[%i]: %f %f\n",i,x_vals[i],y_vals[i]);
	}

	gnuplot_resetplot(plot);

	gnuplot_plot_xy(plot,x_vals,y_vals,nptcls,NULL);


	free(x_vals);
	free(y_vals);
	free(px_t);
	free(vx_t);
	free(ix_t);
}

void ParticleListGPU::CPUfree()
{
	// Allocate realkind arrays
	for(int i=0;i<ParticleList_nfloats;i++)
	{
		CUDA_SAFE_CALL(hipFree(*get_float(i)));
	}

	// Allocate int arrays
	for(int i=0;i<ParticleList_nints;i++)
	{
		CUDA_SAFE_CALL(hipFree(*get_int(i)));
	}

	// allocate short ints for cluster id's
	CUDA_SAFE_CALL(hipFree(cluster_id));
}

long long int ParticleListGPU::push(PlasmaData* pdata, FieldData* fields, HOMoments* moments)
{
	long long int result;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pdata_c),pdata,sizeof(PlasmaData)));
	// Copy field data to the device
//	printf("Copying fields to device\n");
	fields_d->copy_from(fields);

//	printf("Nulling device HOMoments\n");
	moments_d -> set_vals(0);

	// Change location of pdata in moments_d to device
	moments_d->pdata = pdata_d;
	fields_d->pdata = pdata_d;

	// Push the particles
//	printf("Pushing particles on the GPU\n");
	result = push_interface2(pdata,fields_d,moments_d);

	// Tally Charge and S2 Moments


//	printf("More Pushing particles on the GPU\n");
	// Change the location of pdata in moments_d to host
	moments_d->pdata = pdata;
	fields_d->pdata = pdata;
	// Copy HO moments to the host
//	printf("Copying device HOMoments to Host\n");
	moments->copy_from(moments_d);

	return result;
}

template<int nSpatial,int nVel> __global__
void write_cluster_ids(PlasmaData* pdata,
					ParticleListGPU particles,
					int* ifinished,
					int nptcls_check)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int gidx = tidx+blockDim.x*bidx;
	int stride = blockDim.x*gridDim.x;

	int nclusters_x = (pdata->nx+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
	int nclusters_y = (pdata->ny+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
	int nclusters_z = (pdata->nz+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;

	while(gidx < nptcls_check)
	{
		int ix,iy,iz;
		ix = particles.ix[gidx]/pdata->ClusterSortDim;
		iy = particles.iy[gidx]/pdata->ClusterSortDim;
		iz = particles.iz[gidx]/pdata->ClusterSortDim;

		particles.cluster_id[gidx] = ix + nclusters_x*(iy + nclusters_y*iz);
		particles.particleIDs[gidx] = gidx;
		printf("ix,iy,iz[%i] = %i %i %i %i\n",gidx,ix,iy,iz,particles.cluster_id[gidx]);

		gidx += stride;
	}

}


//template<int nSpatial,int nVel,bool iEM> __global__
//void SimpleGPUPush(PlasmaData* 				pdata,
//					FieldDataGPU 			fields,
//					HOMoments 				moments2,
//					ParticleListGPU	particles,
//					int* num_subcycles)
//{
//	int tidx = threadIdx.x;
//	int bidx = blockIdx.x;
//	int gidx = tidx+blockDim.x*bidx;
//	int stride = blockDim.x*gridDim.x;
//
//	int pid = gidx;
//
//	long long int num_subcycles_thread = 0;
//
//	__shared__ realkind currentx[256];
//	__shared__ realkind charge_s[256];
//	__shared__ realkind S2xx_s[256];
//	while(tidx < 256)
//	{
//		currentx[tidx] = 0;
//		charge_s[tidx] = 0;
//		S2xx_s[tidx] = 0;
//		tidx += blockDim.x;
//	}
//	tidx = threadIdx.x;
//
//
//	ParticleListGPUSimple particles2 = particles;
//	FieldDataGPU 			fields2 = fields;
//	FieldData*				fields3 = &fields2;
//	HOMoments 				moments = moments2;
////	PlasmaData pdata = *pdata2;
//
//	ParticleObjNT<1,nSpatial,nVel,iEM> particle(&pid);
//	typevecN<int,1> iter;
//
//	CurrentTally currents(currentx,
//						  &moments.get_val(0,0,0,particles.ispecies,HOMoments_currenty),
//						  &moments.get_val(0,0,0,particles.ispecies,HOMoments_currentz),
//						  make_int3(pdata->nx,pdata->ny,pdata->nz),
//						  pdata->dxdi,pdata->dydi,pdata->dzdi,
//						  pdata->ndimensions);
//
//	ChargeTally charge(charge_s,
//						  make_int3(currents.nx,currents.ny,currents.nz),
//						  currents.dx,currents.dy,currents.dz,
//						  1);
//
//	StressTally stress(S2xx_s,
//			  make_int3(currents.nx,currents.ny,currents.nz),
//			  currents.dx,currents.dy,currents.dz,
//			  1);
//
//	particle.species = particles2.ispecies;
//
//
//	while(pid < particles.nptcls)
//	{
//		iter(0) = 0;
//
//		particle.copy_in_gpu(particles2,0);
//		particle.dt_finished(0) = 0;
//		particle.push(pdata,fields3,&currents,iter,pdata->nSubcycle_max);
//
////		printf("Writing Paerticles Back\n");
//		particle.write_back(particles2,0);
//
//		particles2.num_subcycles[pid] += iter(0);
//
//		charge.tally1d(particles2.px[pid],particles2.ix[pid],1.0);
//		stress.tally1d1v(particles2.px[pid],particles2.vx[pid],particles2.ix[pid],1.0);
//
//		num_subcycles_thread += iter(0);
//
//		pid += stride;
//	}
//
////	printf("exiting gpu shit\n");
//
//	num_subcycles[gidx] = num_subcycles_thread;
//
//	__syncthreads();
//
//	pid = tidx;
//	while(pid < pdata->nx)
//	{
//#ifdef DOUBLE_PRECISION
//		atomicAddD(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_currentx),currentx[pid]);
//		atomicAddD(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_charge),charge_s[pid]);
//		atomicAddD(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_S2xx),S2xx_s[pid]);
//#else
//		atomicAdd(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_currentx),currentx[pid]);
//		atomicAdd(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_charge),charge_s[pid]);
//		atomicAdd(&moments.get_val(pid,0,0,particles.ispecies,HOMoments_S2xx),S2xx_s[pid]);
//#endif
//		pid += blockDim.x;
//	}
//
//}


template<int nSpatial,int nVel,bool iEM> __global__
void GPUBlockPush(PlasmaData* 				pdata,
					FieldDataGPU 			fields_in,
					HOMoments* 				moments,
					ParticleListGPU	particles_in,
					int* num_subcycles,
					int tally_size)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int gidx = tidx+blockDim.x*bidx;
	int stride = blockDim.x*gridDim.x;

	ParticleListGPU particles = particles_in;

	FieldDataGPU 	fields2 = fields_in;
	FieldData* fields = &fields2;


	int pid;

	long long int num_subcycles_thread = 0;

	// Going to shoot for about 4 thread-blocks per cluster,
	int ClustID = bidx/pdata->TBpCluster;
	int ClustID2 = bidx%pdata->TBpCluster;

	if(ClustID > particles_in.nclusters-1)
		return;
	// Check to see if any particles are in the cluster
	if((particles.clusters[ClustID].ifirst < 0)||(particles.clusters[ClustID].ilast < 0))
		return;

	__syncthreads();
	// Allocate shared memory for current tallies
	__shared__ float currentx[33*33];
	__shared__ float currenty[33*33];
	__shared__ float currentz[33*33];




	// Shared copy of the cluster info
	__shared__ ClusterInfo cluster[1];


	// Null out the tallies
	while(tidx < tally_size)
	{
		currentx[tidx] = 0;
		currenty[tidx] = 0;
		currentz[tidx] = 0;
		tidx += blockDim.x;
	}

	tidx = threadIdx.x;
	__syncthreads();

	if(tidx == 0)
		*cluster = particles.clusters[ClustID];

	__syncthreads();

	int ix0c,iy0c,iz0c;
	int ioffset;
	int ix0,iy0,iz0;
	int nx,ny;
	// First figure out the current tally domain
	ioffset = floor((pdata->ClusterStorDim - pdata->ClusterSortDim)/2.0f);
	if(nSpatial == 1)
	{
		ix0c = cluster->clusterid*pdata->ClusterSortDim;
		iy0c = 0;
		iz0c = 0;

		ix0 = ix0c - ioffset;
		iy0 = 0;
		iz0 = 0;
	}
	else if(nSpatial == 2)
	{
		int nclustx = (pdata->nx+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
		iy0c = cluster->clusterid/nclustx;
		ix0c = cluster->clusterid - iy0c*nclustx;
		iy0c *= pdata->ClusterSortDim;
		ix0c *= pdata->ClusterSortDim;

		ix0 = ix0c - ioffset;
		iy0 = iy0c - ioffset;
		iz0 = 0;
	}
	else if(nSpatial == 3)
	{
		int nclusty = (pdata->ny+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
		int nclustx = (pdata->nx+pdata->ClusterSortDim-1)/pdata->ClusterSortDim;
		iz0c = cluster->clusterid/(nclusty*nclustx);
		iy0c = (cluster->clusterid - iz0c*nclustx*nclusty)/nclustx;
		ix0c = cluster->clusterid - (iy0c+iz0c*nclusty)*nclustx;
		iz0c *= pdata->ClusterSortDim;
		iy0c *= pdata->ClusterSortDim;
		ix0c *= pdata->ClusterSortDim;

		ix0 = ix0c - ioffset;
		iy0 = iy0c - ioffset;
		iz0 = iz0c - ioffset;
	}
	CurrentTallyGPU* currents = new CurrentTallyGPU(currentx,currenty,currentz,
			pdata->ClusterStorDim,pdata->ClusterStorDim,pdata->ClusterStorDim,
			ix0,iy0,iz0,pdata->ndimensions);
	// Setup the PExitCheck object


	// Load local CurrentTally Object to shared memory
//		*currents = currents_temp;

	// Now time for the exit condition
	// This guy needs to be 4 smaller than the tally region in order to avoid
	// particles writing outside of the tally region
	ix0 += 2;
	iy0 += 2;
	PExitCheck* exit_checker = new PExitCheckGPU(pdata->dt,pdata->nSubcycle_max,
			ix0,iy0,iz0,
			pdata->ClusterStorDim-4,pdata->ClusterStorDim-4,pdata->ClusterStorDim-4);




	__syncthreads();

	// Each thread figures out its own PID
	pid = cluster->ifirst + ClustID2*blockDim.x + threadIdx.x;
//	printf("First particle in block %i is %i %i\n",bidx,cluster->ifirst+ClustID2*blockDim.x,cluster->ilast);

	// Setup the ParticleObjectNT
	ParticleObjNT<1,nSpatial,nVel,iEM> particle(&pid,exit_checker);


	// Each thread pushes a bunch of particles
	while(pid <= cluster->ilast)
	{
		typevecN<int,1> iter;
		// Load a particle into a ParticleObjNT
		particle.copy_in_gpu(particles,0);

		// Set the subcycle count
		iter(0) = particles.nsubcycles_current[pid];

		// Push that particle
		particle.push(pdata,fields,currents,iter,pdata->nSubcycle_max);

		// ParticleObjNT has exited.
		// Write ParticleObjNT data back to the Particle List
		particle.write_back(particles,0);

		// Save number of subcycles completed
		// for the particle that was just pushed
		particles.nsubcycles_current[pid] = iter(0);

		// Mark particle finished or not for stream compaction array
		if((particle.dt_finished(0) >= pdata->dt)||(iter(0) >= pdata->nSubcycle_max))
			particles.ifinished[pid] = 1;

		printf("Dt finished and iter done[%i] = %f %i\n",pid,particle.dt_finished(0),
				iter(0));


		// increment the number of subcycles this thread has processed
		num_subcycles_thread += iter(0);

		// increment pid and repeat
		pid += 4*blockDim.x;
	}

	// synchronize threads
	__syncthreads();


	// Accumulate currents to global memory
	pid = tidx;
	while(pid < tally_size)
	{
		int ix,iy,iz;

		if(nSpatial == 1)
		{
			ix = pid + currents->ix0;
			iy = 0;
			iz = 0;
		}
		else if(nSpatial == 2)
		{
			iy = pid/currents->nx;
			ix = pid - iy*currents->nx;
			iz = 0;

			ix += currents->ix0;
			iy += currents->iy0;
		}
		else if(nSpatial == 3)
		{
			iz = pid/(currents->nx*currents->ny);
			iy = (pid - currents->nx*currents->ny*iz)/currents->nx;
			ix = pid - currents->nx*(iy+currents->ny*iz);

			ix += currents->ix0;
			iy += currents->iy0;
			iz += currents->iz0;
		}

#ifdef DOUBLE_PRECISION
		atomicAddD(&moments->get_val(ix,iy,iz,particles.ispecies,HOMoments_currentx),currentx[pid]);
		atomicAddD(&moments->get_val(ix,iy,iz,particles.ispecies,HOMoments_currenty),currenty[pid]);
		atomicAddD(&moments->get_val(ix,iy,iz,particles.ispecies,HOMoments_currentz),currentz[pid]);
#else
		atomicAdd(&moments->get_val(ix,iy,iz,particles->ispecies,HOMoments_currentx),currentx[pid]);
		atomicAdd(&moments->get_val(ix,iy,iz,particles->ispecies,HOMoments_currenty),currenty[pid]);
		atomicAdd(&moments->get_val(ix,iy,iz,particles->ispecies,HOMoments_currentz),currentz[pid]);
#endif
		pid += blockDim.x;
	}
}

__global__
void find_cluster_boundaries(ParticleListGPU particles,ClusterInfo* bins,int nptcls_check)
{
	int idx = threadIdx.x;
	int gidx = idx+blockIdx.x*blockDim.x;

	uint binindex;
	uint binindex_left;
	uint binindex_right;

	while(gidx < nptcls_check)
	{
		if(gidx == 0)
		{
			binindex = particles.cluster_id[gidx];
			bins[binindex].ifirst = gidx;
			bins[binindex].clusterid = binindex;
		}
		else if(gidx == nptcls_check-1)
		{
			binindex = particles.cluster_id[gidx];
			bins[binindex].ilast = gidx;
			bins[binindex].clusterid = binindex;
		}

			binindex = particles.cluster_id[gidx];
			binindex_left = particles.cluster_id[max(gidx-1,0)];
			binindex_right = particles.cluster_id[min((gidx+1),(nptcls_check-1))];

			if(binindex_left != binindex)
			{
				printf("ClusterID[%i] = %i\n",gidx,binindex);
				bins[binindex].ifirst = gidx;
				bins[binindex].clusterid = binindex;
			}

			if(binindex_right != binindex)
			{
				printf("ClusterID[%i] = %i\n",gidx,binindex);
				bins[binindex].ilast = gidx;
				bins[binindex].clusterid = binindex;
			}






		gidx += blockDim.x*gridDim.x;
	}
}

__global__
void InitClusters(ClusterInfo* bins,
		int nbins)
{
	int gidx = threadIdx.x+blockDim.x*blockIdx.x;

	// Need to set ifirst, ilast to -1, so that once everything
	// is populated, the bins with no particles have -1 as ifirst and ilast

	while(gidx < nbins)
	{
		bins[gidx].ifirst = -1;
		bins[gidx].ilast = -1;



		gidx += blockDim.x*gridDim.x;
	}


}

__global__
void FieldCheck_g(FieldDataGPU fields_in)
{
	int gidx = threadIdx.x+blockDim.x*blockIdx.x;
	int gidy = threadIdx.y+blockDim.y*blockIdx.y;

	FieldDataGPU fieldst = fields_in;
	FieldData* fields = &fieldst;

	while(gidy < fields->ny)
	{
		while(gidx < fields->nx)
		{
			realkind Bx,By,Bz,Ex,Ey,Ez;

			Bx = fields->getB(gidx,gidy,0,0);
			By = fields->getB(gidx,gidy,0,1);
			Bz = fields->getB(gidx,gidy,0,2);
			Ex = fields->getE(gidx,gidy,0,0);
			Ey = fields->getE(gidx,gidy,0,1);
			Ez = fields->getE(gidx,gidy,0,2);

			printf("Field valas[%i %i] = %e, %e, %e, %e, %e, %e\n",gidx,gidy,Bx,By,Bz,Ex,Ey,Ez);

			gidx += blockDim.x*gridDim.x;
		}

		gidy += blockDim.y*gridDim.y;
	}

}

void FieldCheck(FieldDataGPU fields)
{
	dim3 blockSize(8,8,1);
	dim3 gridSize(1,1,1);

	CUDA_SAFE_KERNEL((FieldCheck_g<<<blockSize,gridSize>>>(fields)))

}

void ParticleListGPU::SetupBlockingInfo(int nptcls_check)
{
	int blockSize = 256;
	int gridSize = (nclusters+blockSize-1)/blockSize;

	// First we need to null out all of the blocking information
	// set first and last id to 0, and nptcls_cluster = 0 for
	// each cluster
	CUDA_SAFE_KERNEL((InitClusters<<<blockSize,gridSize>>>
			(clusters,nclusters)));


	// Now we need to find the first and last particles in each cluster
	gridSize = nclusters;
	CUDA_SAFE_KERNEL((find_cluster_boundaries<<<blockSize,gridSize>>>
			(*this,clusters,nptcls_check)));

}


void ParticleListGPU::ReorderData(int* particleIDs,int nptcls_left_old)
{
	if(sizeof(realkind) == 8)
	{
		// use 64 bit swap

		for(int i=0;i<8;i++)
		{
			realkind* idata = *get_float(i);
			realkind* odata = buffer64;

			ReOrderData64_GPU(idata,odata,particleIDs,nptcls_left_old);

			*get_float(i) = odata;
			buffer64 = idata;
		}

	}
	else
	{
		for(int i=0;i<8;i++)
		{
			realkind* idata = *get_float(i);
			realkind* odata = (realkind*)buffer32;

			ReOrderData32_GPU(idata,odata,particleIDs,nptcls_left_old);

			*get_float(i) = odata;
			buffer32 = (int*)idata;
		}
	}

	// Do the ints
	for(int i=0;i<4;i++)
	{
		int* idata = *get_int(i);
		int* odata = buffer32;

		ReOrderData32_GPU(idata,odata,particleIDs,nptcls_left_old);

		*get_int(i) = odata;
		buffer32 = idata;
	}

	int* idata = nsubcycles_current;
	int* odata = buffer32;

	ReOrderData32_GPU(idata,odata,particleIDs,nptcls_left_old);

	nsubcycles_current = odata;
	buffer32 = idata;

	idata = particleIDs_original;
	odata = buffer32;

	ReOrderData32_GPU(idata,odata,particleIDs,nptcls_left_old);

	particleIDs_original = odata;
	buffer32 = idata;
}



template<int nSpatial,int nVel,bool iEM>
void ParticleListGPU::SortedGPUPushH(PlasmaData* 			pdata,
					FieldDataGPU* 			fields,
					HOMoments* 				moments)
{


	int blockSize = 512;
	int gridSize = 96;
	int tally_size = pdata->ClusterStorDim*pdata->ClusterStorDim;

	FieldDataGPU fields_t = *fields;

	// Make sure everything is in constant memory
	CUDA_SAFE_CALL(hipMemcpyAsync(fields_d_cp,fields,
			sizeof(FieldDataGPU),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(moments_d_cp,moments,
			sizeof(HOMoments),hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyAsync(this_cp,this,
			sizeof(ParticleListGPU),hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipDeviceSynchronize());

//	FieldCheck(fields_t);


	int nptcls_left = nptcls;
	// Particles should initially be sorted due to copy from particles_old

	// Set dt_finished and nsubcycles_current = 0
	CUDA_SAFE_CALL(hipMemset(dt_finished,0,nptcls*sizeof(realkind)));
	CUDA_SAFE_CALL(hipMemset(nsubcycles_current,0,nptcls*sizeof(int)));

	// Basically while any particles have subcycles left we will keep pushing -> sorting
	while(nptcls_left > 0)
	{
		printf("Nptcls_left in GPU push = %i\n",nptcls_left);
		ifinished = (int*)buffer32;
		CUDA_SAFE_CALL(hipMemset(ifinished,0,nptcls_left*sizeof(int)));
		// Set up the blocking information
		printf("Setting up blocking info\n");
		SetupBlockingInfo(nptcls_left);
		// Push the particles
//		void GPUBlockPush(PlasmaData* 				pdata,
//							FieldDataGPU* 			fields,
//							HOMoments* 				moments,
//							ParticleListGPU	particles,
//							int* num_subcycles,
//							int tally_size)
		int pushBlockSize = 512;
		int pushGridSize = pdata->TBpCluster*nclusters;
		int shared_mem_alloc = tally_size*sizeof(realkind)+sizeof(CurrentTallyGPU)+sizeof(PExitCheckGPU)+sizeof(ClusterInfo);
		printf("Pushing particles on gpu\n");
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
		CUDA_SAFE_KERNEL((GPUBlockPush<nSpatial,nVel,iEM><<<pushBlockSize,pushGridSize>>>(
				pdata_d,fields_t,moments_d_cp,
				*this,nsubcycles_thread,tally_size)));

		// Get partition indices, get the number of particles left
		int nptcls_left_old = nptcls_left;
		nptcls_left = GenPartitionIDs(ifinished,particleIDs,nptcls_left);

		// if no particles are left -> break
		if(nptcls_left <= 0)
			break;

		// Write the cluster indices
		CUDA_SAFE_KERNEL((write_cluster_ids<nSpatial,nVel><<<blockSize,gridSize>>>(
				pdata_d,*this,ifinished,nptcls_left_old)));


		// Partition the cluster indices
		ReOrderData32_GPU(cluster_id,buffer32,particleIDs,nptcls_left_old);
		int* temp = cluster_id;
		cluster_id = buffer32;
		buffer32 = temp;

		// Sort the partition indices using cluster indices as keys (nptcls_left)
		SortByKey(particleIDs,cluster_id,nptcls_left);

		// Reorder the remaining particle data (nptcls_left_old)
		ReorderData(particleIDs,nptcls_left_old);


		// repeat

	}

	// Sort the particles for charge and s2 tallies
	// Write the cluster indices
	CUDA_SAFE_KERNEL((write_cluster_ids<nSpatial,nVel><<<blockSize,gridSize>>>(
			pdata_d,*this,ifinished,nptcls)));


	SortByKey(particleIDs,cluster_id,nptcls);

	ReorderData(particleIDs,nptcls);

	// Setup blocking info
	SetupBlockingInfo(nptcls);
	// Do a charge and s2 tally





//	//printf("GPU Pushing Particles\n");
//	CUDA_SAFE_KERNEL((SimpleGPUPush<nSpatial,nVel,iEM><<<blocksize,gridsize>>>
//			(particles->pdata_d,*fields,*moments,*particles,num_subcycles)));
	//printf("GPU Finished Pushing Particles\n");
}


long long int ParticleListGPU::push_interface2(PlasmaData* pdata,
		FieldDataGPU* fields,
		HOMoments* moments)
{


	int num_threads = gridsize*blocksize;



	// Template Selection
	switch(pdata->ndimensions)
	{
	case 1:
		switch(pdata->nVelocity)
		{
		case 1:
			if(!pdata->iEM)
				SortedGPUPushH<1,1,0>(pdata_d,fields,moments);
			else
				SortedGPUPushH<1,1,1>(pdata,fields,moments);
			break;
		case 2:
			if(pdata->iEM == 0)
				SortedGPUPushH<1,2,0>(pdata,fields,moments);
			else
				SortedGPUPushH<1,2,1>(pdata,fields,moments);
			break;
		case 3:
			if(pdata->iEM == 0)
				SortedGPUPushH<1,3,0>(pdata,fields,moments);
			else
				SortedGPUPushH<1,3,1>(pdata,fields,moments);
			break;
		default:
			break;
		}

		break;
	case 2:
		switch(pdata->nVelocity)
		{
		case 2:
			if(pdata->iEM == 0)
				SortedGPUPushH<2,2,0>(pdata,fields,moments);
			else
				SortedGPUPushH<2,2,1>(pdata,fields,moments);
			break;
		case 3:
			if(pdata->iEM == 0)
				SortedGPUPushH<2,3,0>(pdata,fields,moments);
			else
				SortedGPUPushH<2,3,1>(pdata,fields,moments);
			break;
		default:
			break;
		}

		break;
	case 3:
		switch(pdata->nVelocity)
		{
		case 3:
			if(pdata->iEM == 0)
				SortedGPUPushH<3,3,0>(pdata,fields,moments);
			else
				SortedGPUPushH<3,3,1>(pdata,fields,moments);
			break;
		default:
			break;
		}
		break;

	default:
		break;

	}


//	thrust::device_ptr<int> nsubcycles_t(nsubcycles);


	// Reduce and return the total number of particle-subcycles taken
	long long int result = 0;// = thrust::reduce(nsubcycles_t,nsubcycles_t+num_threads);

	int* nsubcycles_temp = (int*)malloc(gridsize*blocksize*sizeof(int));
	CUDA_SAFE_CALL(hipMemcpy(nsubcycles_temp,nsubcycles_thread,gridsize*blocksize*sizeof(int),hipMemcpyDeviceToHost))

	for(int i=0;i<gridsize*blocksize;i++)
		result += nsubcycles_temp[i];

	free(nsubcycles_temp);

	return result;





}








