#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------*/
/**
  @file		CurrentTally.cu
*/
/*--------------------------------------------------------------------------*/
#include "CurrentTally.h"
#include "ShapeFunctions.h"

#ifdef GPU_CODE
__device__
void atomicAddD(double* address,double value)
{
	unsigned long long oldval, newval, readback;
   oldval = __double_as_longlong(*address);
   newval = __double_as_longlong(__longlong_as_double(oldval) + value);
   while ((readback=atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
	 {
	  oldval = readback;
	  newval = __double_as_longlong(__longlong_as_double(oldval) + value);
	 }
}
#endif


FUNCTION_TYPE
CurrentTally::CurrentTally(realkind* currentx_in,
						   realkind* currenty_in,
						   realkind* currentz_in,
						   int3 dims_in,
						   realkind spacingx,realkind spacingy,realkind spacingz,
						   int ndimensions_in)
{
	currentx = currentx_in;
	currenty = currenty_in;
	currentz = currentz_in;

	nx = dims_in.x;
	ny = dims_in.y;
	nz = dims_in.z;

	dx = spacingx;
	dy = spacingy;
	dz = spacingz;

	ndimensions = ndimensions_in;

}

FUNCTION_TYPE
CurrentTally::CurrentTally(){}

FUNCTION_TYPE
void CurrentTally::tally1d1v(const realkind px,
		 const realkind vx,
		 const int ix_in,
		 const realkind scale)
{
	int ix;
	realkind vol_inv =  scale;



	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = realkind(0.25)*vx*S1_shape(xp)*vol_inv;

#ifdef GPU_CODE
	#ifdef DOUBLE_PRECISION
		atomicAddD(currentx+ix,temp);
	#else
		atomicAdd(currentx+ix,temp);
	#endif
#else
//		if(isnan(temp))
//			printf("Warning NAN current value at %i with %e, %e\n",ix,vx,vol_inv);

		currentx[ix] += temp;
#endif

	}



}

FUNCTION_TYPE
void CurrentTally::tally1d2v(const realkind px,
		 const realkind vx,const realkind vy,
		 const int ix_in,
		 const realkind scale)
{

	int ix;
	realkind vol_inv =  scale;



	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = realkind(0.25)*vx*S1_shape(xp)*vol_inv;

		currentx[ix] += temp;


	}

	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = 0.25*vy*S1_shape(xp)*vol_inv;

		currenty[ix] += temp;


	}



}

__host__ __device__
void CurrentTally::tally1d3v(const realkind px,
		 const realkind vx,const realkind vy,const realkind vz,
		 const int ix_in,
		 const realkind scale)
{

	int ix;
	realkind vol_inv =  scale;



	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = 0.25*vx*S1_shape(xp)*vol_inv;

		currentx[ix] += temp;


	}

	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = 0.25*vy*S1_shape(xp)*vol_inv;

		currenty[ix] += temp;


	}

	for(int i=0;i<2;i++)
	{
		realkind xp;

		realkind temp;

		xp = i - px;

		ix = ix_in + i;


		ix = (((ix&(nx-1)) + nx)&(nx-1));

		temp = 0.25*vz*S1_shape(xp)*vol_inv;

		currentz[ix] += temp;


	}



}

__host__ __device__
void CurrentTally::tally2d2v(const realkind px,const realkind py,
		 const realkind vx,const realkind vy,
		 const int ix_in,const int iy_in,
		 const realkind scale)
{

	int ix,iy;
	realkind vol_inv =  scale;


	// x component

		for(int j=-1;j<2;j++)
		{
			for(int i=0;i<2;i++)
			{
				realkind xp, yp;

				realkind temp;

				xp = i - px;
				yp = j + 0.5 - py;

				ix = ix_in + i;
				iy = iy_in + j;

				ix = ((ix%nx + nx)%nx);
				iy = ((iy%ny + ny)%ny);

				temp = 0.5*vx*S1_shape(xp)*S2_shape(yp)*vol_inv;

				currentx[ix + nx*(iy)] += temp;


			}
		}


		// y component

		for(int j=0;j<2;j++)
		{
			for(int i=-1;i<2;i++)
			{
				realkind xp, yp;

				realkind temp;

				xp = i + 0.5 - px;
				yp = j - py;

				ix = ix_in + i;
				iy = iy_in + j;

				ix = ((ix%nx + nx)%nx);
				iy = ((iy%ny + ny)%ny);

				temp = 0.5f*vy*S2_shape(xp)*S1_shape(yp)*vol_inv;

				currenty[ix + nx*(iy)] += temp;


			}
		}



}

__host__ __device__
void CurrentTally::tally2d3v(const realkind px,const realkind py,
		 const realkind vx,const realkind vy,const realkind vz,
		 const int ix_in,const int iy_in,
		 const realkind scale)
{

	int ix,iy;
	realkind vol_inv =  scale;


	// x component

		for(int j=-1;j<2;j++)
		{
			for(int i=0;i<2;i++)
			{
				realkind xp, yp;

				realkind temp;

				xp = i - px;
				yp = j + 0.5 - py;

				ix = ix_in + i;
				iy = iy_in + j;

				ix = ((ix%nx + nx)%nx);
				iy = ((iy%ny + ny)%ny);

				temp = 0.5f*vx*S1_shape(xp)*S2_shape(yp)*vol_inv;
//				printf("temp = %f\n",temp);
				currentx[ix + nx*(iy)] += temp;


			}
		}


		// y component

		for(int j=0;j<2;j++)
		{
			for(int i=-1;i<2;i++)
			{
				realkind xp, yp;

				realkind temp;

				xp = i + 0.5 - px;
				yp = j - py;

				ix = ix_in + i;
				iy = iy_in + j;

				ix = ((ix%nx + nx)%nx);
				iy = ((iy%ny + ny)%ny);

				temp = 0.5f*vy*S2_shape(xp)*S1_shape(yp)*vol_inv;

				currenty[ix + nx*(iy)] += temp;


			}
		}


		// z component
		for(int j=-1;j<2;j++)
		{
			for(int i=-1;i<2;i++)
			{
				realkind xp, yp;

				realkind temp;

				xp = i + 0.5 - px;
				yp = j + 0.5 - py;

				ix = ix_in + i;
				iy = iy_in + j;


				ix = ((ix%nx + nx)%nx);
				iy = ((iy%ny + ny)%ny);


				temp = 0.5f*vz*S2_shape(xp)*S2_shape(yp)*vol_inv;

				currentz[ix + nx*(iy)] += temp;


			}
		}



}

__host__ __device__
void CurrentTally::tally3d3v(const realkind px,const realkind py,const realkind pz,
		 const realkind vx,const realkind vy,const realkind vz,
		 const int ix,const int iy,const int iz,
		 const realkind scale)
{

}

__host__ __device__
void CurrentTally::tally(const realkind px, const realkind py, const realkind pz,
						 const realkind vx, const realkind vy, const realkind vz,
						 const int ix_in, const int iy_in, const int iz_in,
						 const realkind scale)
{

	switch(ndimensions)
	{
	case 1:
		tally1d1v(px,vx,ix_in,scale);
		break;
	case 2:
		tally2d3v(px,py,vx,vy,vz,ix_in,iy_in,scale);
		break;
	case 3:
		tally3d3v(px,py,pz,vx,vy,vz,
				ix_in,iy_in,iz_in,scale);
		break;
	default:
		break;
	}

//	int ix,iy,iz;
//	realkind vol_inv =  scale;
//	if(ndimensions == 1)
//	{
//
//		for(int i=0;i<2;i++)
//		{
//			realkind xp;
//
//			realkind temp;
//
//			xp = i - px;
//
//			ix = ix_in + i;
//
//
//			ix = (((ix%(nx)) + nx)%(nx));
//
//			temp = 0.25*vx*S1_shape(xp)*vol_inv;
//
//			currentx[ix] += temp;
//
//
//		}
//
//
//
//	}
//	else
//	{
//
//		// x component
//		for(int k=-1;k<2;k++)
//		{
//			for(int j=-1;j<2;j++)
//			{
//				for(int i=0;i<2;i++)
//				{
//					realkind xp, yp, zp;
//
//					realkind temp;
//
//					xp = i - px;
//					yp = j + 0.5 - py;
//					zp = k + 0.5 - pz;
//
//					ix = ix_in + i;
//					iy = iy_in + j;
//					iz = iz_in + k;
//
//					ix = ((ix%nx + nx)%nx);
//					iy = ((iy%ny + ny)%ny);
//					iz = ((iz%nz + nz)%nz);
//
//					temp = vx*S1_shape(xp)*S2_shape(yp)*S2_shape(zp)*vol_inv;
//
//					currentx[ix + nx*(iy + ny*(iz))] += temp;
//
//
//				}
//			}
//		}
//
//		// y component
//		for(int k=-1;k<2;k++)
//		{
//			for(int j=0;j<2;j++)
//			{
//				for(int i=-1;i<2;i++)
//				{
//					realkind xp, yp, zp;
//
//					realkind temp;
//
//					xp = i + 0.5 - px;
//					yp = j - py;
//					zp = k + 0.5 - pz;
//
//					ix = ix_in + i;
//					iy = iy_in + j;
//					iz = iz_in + k;
//
//					ix = ((ix%nx + nx)%nx);
//					iy = ((iy%ny + ny)%ny);
//					iz = ((iz%nz + nz)%nz);
//
//					temp = vy*S2_shape(xp)*S1_shape(yp)*S2_shape(zp)*vol_inv;
//
//					currenty[ix + nx*(iy + ny*(iz))] += temp;
//
//
//				}
//			}
//		}
//
//		// z component
//		for(int k=0;k<2;k++)
//		{
//			for(int j=-1;j<2;j++)
//			{
//				for(int i=-1;i<2;i++)
//				{
//					realkind xp, yp, zp;
//
//					realkind temp;
//
//					xp = i + 0.5 - px;
//					yp = j + 0.5 - py;
//					zp = k - pz;
//
//					ix = ix_in + i;
//					iy = iy_in + j;
//					iz = iz_in + k;
//
//					ix = ((ix%nx + nx)%nx);
//					iy = ((iy%ny + ny)%ny);
//					iz = ((iz%nz + nz)%nz);
//
//					temp = vz*S2_shape(xp)*S2_shape(yp)*S1_shape(zp)*vol_inv;
//
//					currentz[ix + nx*(iy + ny*(iz))] += temp;
//
//
//				}
//			}
//		}
//	}


}
