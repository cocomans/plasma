#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------*/
/**
  @file		HOMoments.cu
*/
/*--------------------------------------------------------------------------*/
#include "HOMoments.h"

#include "ParallelInfo.h"
#include "math.h"
#include <omp.h>

HOMoments::HOMoments(PlasmaData* pdata_in,int device_type_in)
{

	pdata = pdata_in;

	device_type = device_type_in;

	int nx = pdata->nx;
	int ny = pdata->ny;
	int nz = pdata->nz;

	int ntotal = nx*ny*nz*pdata->nspecies;

	if(device_type == 0)
	{
		charge 		= (realkind*)malloc(ntotal*sizeof(realkind));

		currentx 	= (realkind*)malloc(ntotal*sizeof(realkind));
		currenty 	= (realkind*)malloc(ntotal*sizeof(realkind));
		currentz 	= (realkind*)malloc(ntotal*sizeof(realkind));

		S2xx = (realkind*)malloc(ntotal*sizeof(realkind));
		S2xy = (realkind*)malloc(ntotal*sizeof(realkind));
		S2xz = (realkind*)malloc(ntotal*sizeof(realkind));
		S2yy = (realkind*)malloc(ntotal*sizeof(realkind));
		S2yz = (realkind*)malloc(ntotal*sizeof(realkind));
		S2zz = (realkind*)malloc(ntotal*sizeof(realkind));

	}
	else if(device_type == 1)
	{
#ifndef NO_CUDA
		CUDA_SAFE_CALL(hipMalloc((void**)&charge,ntotal*sizeof(realkind)));

		CUDA_SAFE_CALL(hipMalloc((void**)&currentx,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&currenty,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&currentz,ntotal*sizeof(realkind)));

		CUDA_SAFE_CALL(hipMalloc((void**)&S2xx,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&S2xy,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&S2xz,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&S2yy,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&S2yz,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMalloc((void**)&S2zz,ntotal*sizeof(realkind)));
#endif
	}

	set_vals(0);
}

void HOMoments::set_vals(realkind val_in)
{
	int nx = pdata->nx;
	int ny = pdata->ny;
	int nz = pdata->nz;

	int ntotal = nx*ny*nz*pdata->nspecies;

	if(device_type == 0)
	{

		memset(charge,0,ntotal*sizeof(realkind));

		memset(currentx,0,ntotal*sizeof(realkind));
		memset(currenty,0,ntotal*sizeof(realkind));
		memset(currentz,0,ntotal*sizeof(realkind));

		memset(S2xx,0,ntotal*sizeof(realkind));
		memset(S2xy,0,ntotal*sizeof(realkind));
		memset(S2xz,0,ntotal*sizeof(realkind));
		memset(S2yy,0,ntotal*sizeof(realkind));
		memset(S2yz,0,ntotal*sizeof(realkind));
		memset(S2zz,0,ntotal*sizeof(realkind));

//		for(int l=0;l<pdata->nspecies;l++)
//		{
//
//			for(int k=0;k<pdata->nz;k++)
//			{
//
//				for(int j=0;j<pdata->ny;j++)
//				{
//					for(int i=0;i<pdata->nx;i++)
//					{
//						int id_out = i+pdata->nx*(j+pdata->ny*(k+pdata->nz*l));
//
//						charge[id_out] = val_in;
//
//						currentx[id_out] = val_in;
//						currenty[id_out] = val_in;
//						currentz[id_out] = val_in;
//
//						S2xx[id_out] = val_in;
//						S2xy[id_out] = val_in;
//						S2xz[id_out] = val_in;
//						S2yy[id_out] = val_in;
//						S2yz[id_out] = val_in;
//						S2zz[id_out] = val_in;
//					}
//				}
//			}
//		}
	}
	else if(device_type == 1)
	{

#ifndef NO_CUDA
		CUDA_SAFE_CALL(hipMemset(charge,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(currentx,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(currenty,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(currentz,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2xx,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2xy,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2xz,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2yy,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2yz,val_in,ntotal*sizeof(realkind)));
		CUDA_SAFE_CALL(hipMemset(S2zz,val_in,ntotal*sizeof(realkind)));

#endif

	}
}

void HOMoments::apply_weights(void)
{

	omp_set_num_threads(pdata->num_cores);

	int nx = pdata->nx;
	int ny = pdata->ny;
	int nz = pdata->nz;

	int ntotal = nx*ny*nz*pdata->nspecies;

	realkind dv = pdata->didx*pdata->didy*pdata->didz;
	for(int l=0;l<pdata->nspecies;l++)
	{
		for(int k=0;k<pdata->nz;k++)
		{
#pragma omp for
			for(int j=0;j<pdata->ny;j++)
			{
				for(int i=0;i<pdata->nx;i++)
				{
					int id_out =  i+pdata->nx*(j+pdata->ny*(k+pdata->nz*l));

					charge[id_out] *= pdata->wspecies[l]*dv;

					currentx[id_out] *= pdata->wspecies[l]*dv;
					currenty[id_out] *= pdata->wspecies[l]*dv;
					currentz[id_out] *= pdata->wspecies[l]*dv;

					S2xx[id_out] *= pdata->wspecies[l]*dv;
					S2xy[id_out] *= pdata->wspecies[l]*dv;
					S2xz[id_out] *= pdata->wspecies[l]*dv;
					S2yy[id_out] *= pdata->wspecies[l]*dv;
					S2yz[id_out] *= pdata->wspecies[l]*dv;
					S2zz[id_out] *= pdata->wspecies[l]*dv;
				}
			}
		}
	}

	realkind* temp = (realkind*)malloc(ntotal*sizeof(realkind));

	if(pdata->ndimensions == 1)
		for(int l=0;l<pdata->nspecies;l++)
		{
			for(int k=0;k<pdata->nz;k++)
			{
#pragma omp for
				for(int j=0;j<pdata->ny;j++)
				{
					for(int m=0;m<5;m++)
					{
						enum HOMoments_moment moment;
						if(m==0)
							moment = HOMoments_currentx;
						else if(m==1)
							moment = HOMoments_currenty;
						else if(m==2)
							moment = HOMoments_currentz;
						else if(m==3)
							moment = HOMoments_charge;
						else if(m==4)
							moment = HOMoments_S2xx;

						for(int i=0;i<pdata->nx;i++)
						{
							temp[i] = 0.25*(get_val(i+1,j,k,l,moment)
									+ 2.0*get_val(i,j,k,l,moment)
									+ get_val(i-1,j,k,l,moment));
						}

						for(int i=0;i<pdata->nx;i++)
						{
							get_val(i,j,k,l,moment) = temp[i];
						}
					}
				}
			}
		}

	free(temp);


}

realkind HOMoments::evaluate_energy(void)
{
	double energy = 0.0;
	for(int l=0;l<pdata->nspecies;l++)
	{
		for(int k=0;k<pdata->nz;k++)
		{
			for(int j=0;j<pdata->ny;j++)
			{
				for(int i=0;i<pdata->nx;i++)
				{
					int id_out =  i+pdata->nx*(j+pdata->ny*(k+pdata->nz*l));

					energy += (S2xx[id_out]+S2yy[id_out]+S2zz[id_out])*pdata->mspecies[l];
				}
			}
		}
	}

		energy = 0.5*energy*pdata->dxdi*pdata->dydi*pdata->dzdi;

	return energy;
}

__host__
void HOMoments::reduce(int tid)
{
//	printf("Reducing HO Moments\n");
	int nthreads = omp_get_num_threads();
	HOMoments* root_moment = this-tid;

	int ntotal = pdata->nx * pdata->ny * pdata->nz * pdata->nspecies;

	int stride = (ntotal + nthreads - 1)/nthreads;

	int istart = tid * stride;
	int iend = std::min((tid + 1) * stride-1,ntotal-1);

	//int ndo = iend - istart + 1;
#pragma omp barrier

	for(int j=1;j<nthreads;j++)
	{
		for(int i=istart;i<=iend;i++)
		{
			root_moment->charge[i] += (root_moment+j)->charge[i];

			root_moment->currentx[i] += (root_moment+j)->currentx[i];
			root_moment->currenty[i] += (root_moment+j)->currenty[i];
			root_moment->currentz[i] += (root_moment+j)->currentz[i];

			root_moment->S2xx[i] += (root_moment+j)->S2xx[i];
			root_moment->S2xy[i] += (root_moment+j)->S2xy[i];
			root_moment->S2xz[i] += (root_moment+j)->S2xz[i];
			root_moment->S2yy[i] += (root_moment+j)->S2yy[i];
			root_moment->S2yz[i] += (root_moment+j)->S2yz[i];
			root_moment->S2zz[i] += (root_moment+j)->S2zz[i];
		}

	}

#pragma omp barrier

	//printf("Finished Reducing HO Moments\n");

}

void HOMoments::mpi_reduce(HOMoments** all_moments,ParallelInfo* pll_info)
{
	//printf("MPI Reducing HO Moments\n");
	int nthreads = pll_info->nthreads;
	int tid = pll_info->tid;
	HOMoments* root_moment = all_moments[1];
	HOMoments* mpi_root_moment = all_moments[0];

	int ntotal = pdata->nx * pdata->ny * pdata->nz * pdata-> nspecies;

	int stride = (ntotal + nthreads - 1)/nthreads;

	int istart = tid * stride;
	int iend = std::min((tid + 1) * stride-1,ntotal-1);

	//int ndo = iend - istart + 1;
/*	for(int i=istart;i<iend;i++)
	{
		for(int j=1;j<nthreads;j++)
		{
			root_moment->charge[i] += all_moments[j+1]->charge[i];

			root_moment->currentx[i] += all_moments[j+1]->currentx[i];
			root_moment->currenty[i] += all_moments[j+1]->currenty[i];
			root_moment->currentz[i] += all_moments[j+1]->currentz[i];

			root_moment->S2[i] += all_moments[j+1]->S2[i];
		}

	}

*/

	if(pdata->device_type == 0){
	int tid2;
	omp_set_num_threads(pdata->num_cores);
#pragma omp parallel private(tid,nthreads,stride) default(shared) num_threads(pdata->num_cores)
	{
		tid2 = omp_get_thread_num();
		(root_moment+tid2)->reduce(tid2);

	}
	}


//	MPI_Barrier(MPI_COMM_WORLD);

	// Need to do an MPI reduce
	if(tid == 0)
	{
		for(int i=0;i<pdata->nspecies;i++)
		{
			if(pdata->ndimensions == 1)
				ntotal = pdata->nx;
			if(pdata->ndimensions == 2)
				ntotal = pdata->nx*pdata->ny;
			// Reduce Charge
			MPI_Allreduce(&get_val(0,0,0,i,HOMoments_charge),&mpi_root_moment->get_val(0,0,0,i,HOMoments_charge),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);

			// Reduce Current
			MPI_Allreduce(&get_val(0,0,0,i,HOMoments_currentx),&mpi_root_moment->get_val(0,0,0,i,HOMoments_currentx),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 1)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_currenty),&mpi_root_moment->get_val(0,0,0,i,HOMoments_currenty),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 2)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_currentz),&mpi_root_moment->get_val(0,0,0,i,HOMoments_currentz),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);

			MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2xx),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2xx),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 1)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2xy),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2xy),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 2)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2xz),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2xz),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 1)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2yy),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2yy),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 2)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2yz),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2yz),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
			if(pdata->nVelocity > 2)MPI_Allreduce(&get_val(0,0,0,i,HOMoments_S2zz),&mpi_root_moment->get_val(0,0,0,i,HOMoments_S2zz),ntotal,MPI_REALKIND,MPI_SUM,MPI_COMM_WORLD);
		}
	}

	if((pll_info->myid_mpi == 0)||(pdata->lo_all))
	{

		if(pdata->ndimensions == 1)
		{
			for(int l=0;l<pdata->nspecies;l++)
			{
				for(int k=0;k<pdata->nz;k++)
				{
					for(int j=0;j<pdata->ny;j++)
					{
						for(int i=0;i<pdata->nx;i++)
						{
							int id_out =  i+pdata->nx*(j+pdata->ny*(k+pdata->nz*l));
							int id_in =  i+pdata->nx*(0+pdata->ny*(0+pdata->nz*l));

							mpi_root_moment->charge[id_out] = mpi_root_moment->charge[id_in];

							mpi_root_moment->currentx[id_out] = mpi_root_moment->currentx[id_in];
							mpi_root_moment->currenty[id_out] = mpi_root_moment->currenty[id_in];
							mpi_root_moment->currentz[id_out] = mpi_root_moment->currentz[id_in];

							mpi_root_moment->S2xx[id_out] = mpi_root_moment->S2xx[id_in];
							mpi_root_moment->S2xy[id_out] = mpi_root_moment->S2xy[id_in];
							mpi_root_moment->S2xz[id_out] = mpi_root_moment->S2xz[id_in];
							mpi_root_moment->S2yy[id_out] = mpi_root_moment->S2yy[id_in];
							mpi_root_moment->S2yz[id_out] = mpi_root_moment->S2yz[id_in];
							mpi_root_moment->S2zz[id_out] = mpi_root_moment->S2zz[id_in];
						}
					}
				}
			}
		}

		if(pdata->ndimensions == 2)
		{
			for(int l=0;l<pdata->nspecies;l++)
			{
				for(int k=0;k<pdata->nz;k++)
				{
					for(int j=0;j<pdata->ny;j++)
					{
						for(int i=0;i<pdata->nx;i++)
						{
							int id_out =  i+pdata->nx*(j+pdata->ny*(k+pdata->nz*l));
							int id_in =  i+pdata->nx*(j+pdata->ny*(0+pdata->nz*l));

							mpi_root_moment->charge[id_out] = mpi_root_moment->charge[id_in];

							mpi_root_moment->currentx[id_out] = mpi_root_moment->currentx[id_in];
							mpi_root_moment->currenty[id_out] = mpi_root_moment->currenty[id_in];
							mpi_root_moment->currentz[id_out] = mpi_root_moment->currentz[id_in];

							mpi_root_moment->S2xx[id_out] = mpi_root_moment->S2xx[id_in];
							mpi_root_moment->S2xy[id_out] = mpi_root_moment->S2xy[id_in];
							mpi_root_moment->S2xz[id_out] = mpi_root_moment->S2xz[id_in];
							mpi_root_moment->S2yy[id_out] = mpi_root_moment->S2yy[id_in];
							mpi_root_moment->S2yz[id_out] = mpi_root_moment->S2yz[id_in];
							mpi_root_moment->S2zz[id_out] = mpi_root_moment->S2zz[id_in];
						}
					}
				}
			}
		}
	}

//	printf("Finished Reducing HO Moments with n_nodes = %i\n",pll_info->n_nodes);


}

void HOMoments::copy_from(HOMoments* src)
{
	// Copy all the moment values from src to this
	int nalloc = pdata->nx*pdata->ny*pdata->nz*pdata->nspecies;

	if((device_type == 0)&&(src->device_type == 0))
	{
		memcpy(charge,src->charge,nalloc*sizeof(realkind));
		memcpy(currentx,src->currentx,nalloc*sizeof(realkind));
		memcpy(currenty,src->currenty,nalloc*sizeof(realkind));
		memcpy(currentz,src->currentz,nalloc*sizeof(realkind));

		memcpy(S2xx,src->S2xx,nalloc*sizeof(realkind));
		memcpy(S2xy,src->S2xy,nalloc*sizeof(realkind));
		memcpy(S2xz,src->S2xz,nalloc*sizeof(realkind));
		memcpy(S2yy,src->S2yy,nalloc*sizeof(realkind));
		memcpy(S2yz,src->S2yz,nalloc*sizeof(realkind));
		memcpy(S2zz,src->S2zz,nalloc*sizeof(realkind));
	}
	else
	{

#ifndef NO_CUDA
		enum hipMemcpyKind kind;

		if(device_type == 1)
		{
			if(src->device_type == 1)

				kind = hipMemcpyDeviceToDevice;
			else if(src->device_type == 0)
				kind = hipMemcpyHostToDevice;
		}
		else
			kind = hipMemcpyDeviceToHost;

		CUDA_SAFE_CALL(hipMemcpy(charge,src->charge,nalloc*sizeof(realkind),kind));

		CUDA_SAFE_CALL(hipMemcpy(currentx,src->currentx,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(currenty,src->currenty,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(currentz,src->currentz,nalloc*sizeof(realkind),kind));

		CUDA_SAFE_CALL(hipMemcpy(S2xx,src->S2xx,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(S2xy,src->S2xy,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(S2xz,src->S2xz,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(S2yy,src->S2yy,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(S2yz,src->S2yz,nalloc*sizeof(realkind),kind));
		CUDA_SAFE_CALL(hipMemcpy(S2zz,src->S2zz,nalloc*sizeof(realkind),kind));
#endif
	}





}

realkind HOMoments::check_charge(HOMoments* moments_old)
{
	double charge_cons = 0;

	int j=0;
	int k=0;

	realkind dv = pdata->didx*pdata->didy*pdata->didz;

//	for(int k=0;k<pdata->nz-1;k++)
//	{
//		for(int j=0;j<pdata->ny-1;j++)
//		{
			for(int i=0;i<pdata->nx;i++)
			{
				realkind deltaj  = 0;
				realkind deltan = 0;

				for(int l=0;l<pdata->nspecies;l++)
				{
					deltaj += (get_val(i+1,j,k,l,HOMoments_currentx)-get_val(i,j,k,l,HOMoments_currentx))*pdata->didx;
					deltan += (get_val(i,j,k,l,HOMoments_charge)-moments_old->get_val(i,j,k,l,HOMoments_charge))/(pdata->dt);
				}

			//	printf("charge_cons(%i) = %e + %e = %e\n",i,deltaj,deltan,deltaj+deltan);

				charge_cons += fabs(deltan + deltaj);
			}
//		}
//	}


	return fabs(charge_cons)/(pdata->nx * pdata->ny *pdata->nz);
}

/*
void HOMoments::reduce(HOMoments** all_moments,ParallelInfo* pll_info)
{
	//printf("Reducing HO Moments\n");
	int nthreads = pll_info->nthreads;
	int tid = pll_info->tid;
	HOMoments* root_moment = all_moments[1];
	HOMoments* mpi_root_moment = all_moments[0];

	int ntotal = pdata->nx * pdata->ny * pdata->nz;

	int stride = (ntotal + nthreads - 1)/nthreads;

	int istart = tid * stride;
	int iend = fmin((tid + 1) * stride-1,ntotal-1);

	//int ndo = iend - istart + 1;
#pragma omp barrier

	for(int i=istart;i<iend;i++)
	{
		for(int j=1;j<nthreads;j++)
		{
			root_moment->charge[i] += all_moments[j+1]->charge[i];

			root_moment->currentx[i] += all_moments[j+1]->currentx[i];
			root_moment->currenty[i] += all_moments[j+1]->currenty[i];
			root_moment->currentz[i] += all_moments[j+1]->currentz[i];

			root_moment->S2[i] += all_moments[j+1]->S2[i];
		}

	}

#pragma omp barrier

	printf("Finished Reducing HO Moments with n_nodes = %i\n",pll_info->n_nodes);

	if(pll_info->n_nodes > 1)
	{
		// Need to do an MPI reduce
		if(tid == 0)
		{
			// Reduce Charge
			MPI_Allreduce(charge,mpi_root_moment->charge,ntotal,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);

			// Reduce Current
			MPI_Allreduce(currentx,mpi_root_moment->currentx,ntotal,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
			MPI_Allreduce(currenty,mpi_root_moment->currenty,ntotal,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
			MPI_Allreduce(currentz,mpi_root_moment->currentz,ntotal,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
		}
	}
	else
	{
		for(int i=istart;i<iend;i++)
		{

			mpi_root_moment->charge[i] = root_moment->charge[i];

			mpi_root_moment->currentx[i] = root_moment->currentx[i];
			mpi_root_moment->currenty[i] = root_moment->currenty[i];
			mpi_root_moment->currentz[i] = root_moment->currentz[i];

			mpi_root_moment->S2[i] = root_moment->S2[i];

		}
	}
#pragma omp barrier

}

*/
__attribute__((noinline))
realkind& HOMoments::get_val(const int ix, const int iy, const int iz,
		const int ispecies,enum HOMoments_moment moment)
{
	realkind* result;

	int ix2,iy2,iz2;


	ix2 = ((ix%pdata->nx)+pdata->nx)%pdata->nx;
	iy2 = ((iy%pdata->ny)+pdata->ny)%pdata->ny;
	iz2 = ((iz%pdata->nz)+pdata->nz)%pdata->nz;


	int iout = ix2 + pdata->nx * (iy2 + pdata->ny * (iz2 + pdata->nz * ispecies));
	switch(moment)
	{
	case HOMoments_charge:
		result = charge + iout;
		break;
	case HOMoments_currentx:
		result = currentx + iout;
		break;
	case HOMoments_currenty:
		result = currenty + iout;
		break;
	case HOMoments_currentz:
		result = currentz + iout;
		break;
	case HOMoments_S2xx:
		result = S2xx + iout;
		break;
	case HOMoments_S2xy:
		result = S2xy + iout;
		break;
	case HOMoments_S2xz:
		result = S2xz + iout;
		break;
	case HOMoments_S2yy:
		result = S2yy + iout;
		break;
	case HOMoments_S2yz:
		result = S2yz + iout;
		break;
	case HOMoments_S2zz:
		result = S2zz + iout;
		break;
	default:
		break;
	}

	return *result;
}

__attribute__((noinline))
const realkind& HOMoments::get_val(const int ix, const int iy, const int iz,
		const int ispecies,enum HOMoments_moment moment)
const
{
	realkind* result;

	int ix2,iy2,iz2;

	ix2 = ((ix%pdata->nx)+pdata->nx)%pdata->nx;
	iy2 = ((iy%pdata->ny)+pdata->ny)%pdata->ny;
	iz2 = ((iz%pdata->nz)+pdata->nz)%pdata->nz;


	int iout = ix2 + pdata->nx * (iy2 + pdata->ny * (iz2 + pdata->nz * ispecies));
	switch(moment)
	{
	case HOMoments_charge:
		result = charge + iout;
		break;
	case HOMoments_currentx:
		result = currentx + iout;
		break;
	case HOMoments_currenty:
		result = currenty + iout;
		break;
	case HOMoments_currentz:
		result = currentz + iout;
		break;
	case HOMoments_S2xx:
		result = S2xx + iout;
		break;
	case HOMoments_S2xy:
		result = S2xy + iout;
		break;
	case HOMoments_S2xz:
		result = S2xz + iout;
		break;
	case HOMoments_S2yy:
		result = S2yy + iout;
		break;
	case HOMoments_S2yz:
		result = S2yz + iout;
		break;
	case HOMoments_S2zz:
		result = S2zz + iout;
		break;
	default:
		break;
	}

	return *result;
}

void HOMoments::init_plot()
{
	plot_handle = gnuplot_init();
	//gnuplot_cmd(plot_handle,"set zrange [-0.2:0.2]");
	//gnuplot_cmd(plot_handle,"set cbrange [-0.1:0.1]");
}

void HOMoments::reset_plot()
{
	gnuplot_resetplot(plot_handle);
	//gnuplot_cmd(plot_handle,"set zrange [-0.2:0.2]");
}

void HOMoments::close_plot()
{
	gnuplot_close(plot_handle);
}

//void HOMoments::currentplot(int position,
//		int plane = 0,
//		int ispecies = 0)
//{
//	/*
//	 * plane = 0: xy plane
//	 * plane = 1: xz plane
//	 * plane = 2: yz plane
//	 */
//	int nx, ny;
//	int i,j,k;
//
//	int* i_out_t;
//	int* j_out_t;
//
//	float dx,dy;
//	float x0,y0;
//
//	float* x_vals;
//	float* y_vals;
//	float* z_vals;
//
//	realkind* vals_in;
//	char* title;
//
//	if(plane == 0)
//	{
//		nx = pdata->nx;
//		ny = pdata->ny;
//
//		dx = pdata->dxdi;
//		dy = pdata->dydi;
//
//		x0 = pdata->xmin;
//		y0 = pdata->ymin;
//
//		i_out_t = &i;
//		j_out_t = &j;
//
//		gnuplot_cmd(plot_handle,"set xlabel \"x\"");
//		gnuplot_cmd(plot_handle,"set ylabel \"y\"");
//	}
//	else if(plane == 1)
//	{
//		nx = pdata->nx;
//		ny = pdata->nz;
//
//		dx = pdata->dxdi;
//		dy = pdata->dzdi;
//
//		x0 = pdata->xmin;
//		y0 = pdata->zmin;
//
//		i_out_t = &i;
//		j_out_t = &k;
//
//		gnuplot_cmd(plot_handle,"set xlabel \"x\"");
//		gnuplot_cmd(plot_handle,"set ylabel \"z\"");
//	}
//	else if(plane == 2)
//	{
//		nx = pdata->ny;
//		ny = pdata->nz;
//
//		dx = pdata->dydi;
//		dy = pdata->dzdi;
//
//		x0 = pdata->ymin;
//		y0 = pdata->zmin;
//
//		i_out_t = &j;
//		j_out_t = &k;
//
//		gnuplot_cmd(plot_handle,"set xlabel \"y\"");
//		gnuplot_cmd(plot_handle,"set ylabel \"z\"");
//	}
//
//	int& i_out = *i_out_t;
//	int& j_out = *j_out_t;
//
//	x_vals = (float*)malloc(nx*ny*sizeof(float));
//	y_vals = (float*)malloc(nx*ny*sizeof(float));
//	z_vals = (float*)malloc(nx*ny*sizeof(float));
//}

void HOMoments::plot(int position, int plane = 0,int ispecies = 0, enum HOMoments_moment moment = HOMoments_charge)
{
	/*
	 * plane = 0: xy plane
	 * plane = 1: xz plane
	 * plane = 2: yz plane
	 */
	int nx, ny;
	int i,j,k;

	int* i_out_t;
	int* j_out_t;

	float dx,dy;
	float x0,y0;

	float* x_vals;
	float* y_vals;
	float* z_vals;

	float* dx_vals;
	float* dy_vals;
	float* dz_vals;

	realkind* vals_in;
	char* title;

	switch(moment)
	{
	case HOMoments_charge:
		vals_in = charge;
		title = "Charge Density";
		break;
	case HOMoments_currentx:
		vals_in = currentx;
		title = "Current Density - x";
		break;
	case HOMoments_currenty:
		vals_in = currenty;
		title = "Current Density - y";
		break;
	case HOMoments_currentz:
		vals_in = currentz;
		title = "Current Density - z";
		break;
	case HOMoments_S2xx:
		vals_in = S2xx;
		title = "Stress";
		break;
	case HOMoments_S2xy:
		vals_in = S2xy;
		title = "Stress";
		break;
	case HOMoments_S2xz:
		vals_in = S2xz;
		title = "Stress";
		break;
	case HOMoments_S2yy:
		vals_in = S2yy;
		title = "Stress yy";
		break;
	case HOMoments_S2yz:
		vals_in = S2yz;
		title = "Stress yz";
		break;
	case HOMoments_S2zz:
		vals_in = S2zz;
		title = "Stress zz";
		break;
	case HOMoments_currentxyz:
		title = "Total Current";
		break;
	default:
		break;
	}

	if(plane == 0)
	{
		nx = pdata->nx+1;
		ny = pdata->ny+1;

		dx = pdata->dxdi;
		dy = pdata->dydi;

		x0 = pdata->xmin;
		y0 = pdata->ymin;

		i_out_t = &i;
		j_out_t = &j;

		gnuplot_cmd(plot_handle,"set xlabel \"x\"");
		gnuplot_cmd(plot_handle,"set ylabel \"y\"");
	}
	else if(plane == 1)
	{
		nx = pdata->nx+1;
		ny = pdata->nz+1;

		dx = pdata->dxdi;
		dy = pdata->dzdi;

		x0 = pdata->xmin;
		y0 = pdata->zmin;

		i_out_t = &i;
		j_out_t = &k;

		gnuplot_cmd(plot_handle,"set xlabel \"x\"");
		gnuplot_cmd(plot_handle,"set ylabel \"z\"");
	}
	else if(plane == 2)
	{
		nx = pdata->ny+1;
		ny = pdata->nz+1;

		dx = pdata->dydi;
		dy = pdata->dzdi;

		x0 = pdata->ymin;
		y0 = pdata->zmin;

		i_out_t = &j;
		j_out_t = &k;

		gnuplot_cmd(plot_handle,"set xlabel \"y\"");
		gnuplot_cmd(plot_handle,"set ylabel \"z\"");
	}

	int& i_out = *i_out_t;
	int& j_out = *j_out_t;

	x_vals = (float*)malloc(nx*ny*sizeof(float));
	y_vals = (float*)malloc(nx*ny*sizeof(float));
	z_vals = (float*)malloc(nx*ny*sizeof(float));

	dx_vals = (float*)malloc(nx*ny*sizeof(float));
	dy_vals = (float*)malloc(nx*ny*sizeof(float));
	dz_vals = (float*)malloc(nx*ny*sizeof(float));

	float scale = sqrt(pdata->Lx*pdata->Ly/(pdata->nx*pdata->ny))/2;

	if(plane == 0)
	{
		k = 0;

		for(j=0;j<=pdata->ny;j++)
		{
			for(i=0;i<=pdata->nx;i++)
			{
				x_vals[i_out] = dx*i_out + x0;
				y_vals[j_out] = dy*j_out + y0;

				float temp = 0;

				float xcur = 0;
				float ycur = 0;
				float zcur = 0;

				for(int l=0;l<pdata->nspecies;l++)
				{

					if(moment == HOMoments_currentxyz)
					{
						xcur += get_val(i,j,k,l,HOMoments_currentx)*pdata->qspecies[l];
						ycur += get_val(i,j,k,l,HOMoments_currenty)*pdata->qspecies[l];
						zcur += get_val(i,j,k,l,HOMoments_currentz)*pdata->qspecies[l];
					}
					else
					{
						float t2 = get_val(i,j,k,l,moment)*pdata->qspecies[l];

						if(isnan(t2))
						{
							printf("%s[%i, %i, %i, %i] is nan\n",title,i,j,k,l);
						}
						temp += t2;
					}

				}

				if(moment == HOMoments_currentxyz)
				{

					z_vals[i_out+nx*j_out] = sqrt(xcur*xcur+ycur*ycur+zcur*zcur);
					dx_vals[i_out+nx*j_out] = xcur*scale/z_vals[i_out+nx*j_out];
					dy_vals[i_out+nx*j_out] = ycur*scale/z_vals[i_out+nx*j_out];
					dz_vals[i_out+nx*j_out] = zcur*scale/z_vals[i_out+nx*j_out];

				}
				else
				{
					z_vals[i_out+nx*j_out] = temp;
				}
			}
		}
	}
	else if(plane == 1)
	{
		j = position;

		for(k=0;k<pdata->nz;k++)
		{
			for(i=0;i<pdata->nx;i++)
			{
				x_vals[i_out] = dx*i_out + x0;
				y_vals[j_out] = dy*j_out + y0;
				float temp = get_val(i,j,k,ispecies,moment);

				if(isnan(temp))
				{
					printf("%s[%i, %i, %i] is nan\n",title,i,j,k);
				}

				z_vals[i_out+nx*j_out] = pdata->n0/pdata->nptcls*get_val(i,j,k,ispecies,moment);
			}
		}
	}
	else if(plane == 2)
	{
		i = position;

		for(k=0;k<pdata->nz;k++)
		{
			for(j=0;j<pdata->ny;j++)
			{
				x_vals[i_out] = dx*i_out + x0;
				y_vals[j_out] = dy*j_out + y0;
				float temp = get_val(i,j,k,ispecies,moment);

				if(isnan(temp))
				{
					printf("%s[%i, %i, %i] is nan\n",title,i,j,k);
				}

				z_vals[i_out+nx*j_out] = get_val(i,j,k,ispecies,moment);
			}
		}
	}

	//if(ndimensions == 1)
	if(moment == HOMoments_currentxyz)
	{
		gnuplot_plot_xyz(plot_handle,x_vals,y_vals,z_vals,nx,ny,title);
		gnuplot_cmd(plot_handle,"set view map");
		gnuplot_cmd(plot_handle,"unset hidden3d");
		gnuplot_plot_vector3D(plot_handle,x_vals,y_vals,z_vals,dx_vals,dy_vals,dz_vals,nx,ny,title);

	}
	else
	{
		gnuplot_plot_xyz(plot_handle,x_vals,y_vals,z_vals,nx,ny,title);
	}

	free(x_vals);
	free(y_vals);
	free(z_vals);
	free(dx_vals);
	free(dy_vals);
	free(dz_vals);









}
